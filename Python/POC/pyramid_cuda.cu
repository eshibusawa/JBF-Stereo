
#include <hip/hip_runtime.h>
// This file is part of JBF-Stereo.
// Copyright (c) 2022, Eijiro Shibusawa <phd_kimberlite@yahoo.co.jp>
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR
// ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
// ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

extern "C" __global__ void downSampling(
	float* output,
	hipTextureObject_t texImage,
	int width,
	int height)
{
	const int indexX = blockIdx.x * blockDim.x + threadIdx.x;
	const int indexY = blockIdx.y * blockDim.y + threadIdx.y;
	if ((indexX >= width) || (indexY >= height))
	{
		return;
	}
	const int index = indexX + indexY * width;
    float tmp;
    tmp = fmaf(0.25f, tex2D<float>(texImage, 2 * indexX,     2 * indexY), 0);
    tmp = fmaf(0.25f, tex2D<float>(texImage, 2 * indexX + 1, 2 * indexY), tmp);
    tmp = fmaf(0.25f, tex2D<float>(texImage, 2 * indexX,     2 * indexY + 1), tmp);
    tmp = fmaf(0.25f, tex2D<float>(texImage, 2 * indexX + 1, 2 * indexY + 1), tmp);
    output[index] = tmp;
}

extern "C" __global__ void upSampling(
	float* output,
	hipTextureObject_t texDisparity,
	int width,
	int height)
{
	const int indexX = blockIdx.x * blockDim.x + threadIdx.x;
	const int indexY = blockIdx.y * blockDim.y + threadIdx.y;
	if ((indexX >= width) || (indexY >= height))
	{
		return;
	}
	const int index = indexX + indexY * width;
    output[index] = 2 * tex2D<float>(texDisparity, indexX/2, indexY/2);
}
