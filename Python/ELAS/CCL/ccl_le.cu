
#include <hip/hip_runtime.h>
// This file is part of JBF-Stereo.
// Copyright (c) 2023, Eijiro Shibusawa <phd_kimberlite@yahoo.co.jp>
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR
// ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
// ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

// This file is a modified version of CCL <https://github.com/foota/ccl>, see MIT license below.

// The MIT License (MIT)
// Copyright (c) 2012 Noriyuki Futatsugi
// Permission is hereby granted, free of charge, to any person obtaining a copy of this software and associated documentation files (the "Software"), to deal in the Software without restriction, including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of the Software, and to permit persons to whom the Software is furnished to do so, subject to the following conditions:
// The above copyright notice and this permission notice shall be included in all copies or substantial portions of the Software.
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
// Marathon Match - CCL - Label Equivalence

extern "C" __global__ void init_CCL(int L[], int R[], int N)
{
	int id = blockIdx.x * blockDim.x + blockIdx.y * blockDim.x * gridDim.x + threadIdx.x;
	if (id >= N) return;

	L[id] = R[id] = id;
}

extern "C" __global__ void scanning(short D[], int L[], int R[], int* m, int N, int W, short th)
{
	int id = blockIdx.x * blockDim.x + blockIdx.y * blockDim.x * gridDim.x + threadIdx.x;
	if (id >= N) return;

	short Did = D[id];
	int label = N;
	if (id - W >= 0 && abs(Did - D[id-W]) <= th) label = min(label, L[id-W]);
	if (id + W < N  && abs(Did - D[id+W]) <= th) label = min(label, L[id+W]);
	int r = id % W;
	if (r           && abs(Did - D[id-1]) <= th) label = min(label, L[id-1]);
	if (r + 1 != W  && abs(Did - D[id+1]) <= th) label = min(label, L[id+1]);

	if (label < L[id]) {
		R[L[id]] = label;
		*m = 0xffffffff;
	}
}

extern "C" __global__ void analysis(int L[], int R[], int N)
{
	int id = blockIdx.x * blockDim.x + blockIdx.y * blockDim.x * gridDim.x + threadIdx.x;
	if (id >= N) return;

	int label = L[id];
	int ref;
	if (label == id) {
		do { label = R[ref = label]; } while (ref ^ label);
		R[id] = label;
	}
}

extern "C" __global__ void labeling(int L[], int R[], int N)
{
	int id = blockIdx.x * blockDim.x + blockIdx.y * blockDim.x * gridDim.x + threadIdx.x;
	if (id >= N) return;

	L[id] = R[R[L[id]]];
}
